#include "hip/hip_runtime.h"
#include <hip_runtime.h>
#include <vector>
#include <exception>
#include <stdexcept>
#include <string>
#include <cstring>
#include <cstdio>
#include "bench.h"
#include "buffer.h"
#include "stream.h"
#include "timer.h"

using std::vector;
using std::runtime_error;
using std::string;


// Helper function to "round up" units
// Example 1024 B becomes 1 KiB
static string bytesToUnit(size_t size)
{
    char buffer[1024];
    const char* units[] = { "B  ", "KiB", "MiB", "GiB", "TiB" };
    size_t i = 0, n = sizeof(units) / sizeof(units[0]);

    double csize = (double) size;

    while (i < (n - 1) && csize >= 1024.0)
    {
        csize /= 1024.0;
        ++i;
    }

    snprintf(buffer, sizeof(buffer), "%.2f %s", csize, units[i]);
    return string(buffer);
}


// Helper function to get a string representation of a transfer direction
static string transferDirectionToString(hipMemcpyKind direction)
{
    if (direction == hipMemcpyHostToDevice)
    {
        return string("HtoD");
    }
    if (direction == hipMemcpyDeviceToHost)
    {
        return string("DtoH");
    }

    return string("unknown");
}


// Execute transfers and time them
static void timeTransfers(const vector<TransferSpec>& transferSpecs)
{
    hipError_t err;

    for (const TransferSpec& spec : transferSpecs)
    {
        hipStream_t stream = *spec.stream;

        const void* src = spec.direction == hipMemcpyDeviceToHost ? spec.deviceBuffer.get() : spec.hostBuffer.get();
        void* dst = spec.direction == hipMemcpyDeviceToHost ? spec.hostBuffer.get() : spec.deviceBuffer.get();

        err = hipEventRecord(spec.timer->started, stream);
        if (err != hipSuccess)
        {
            throw runtime_error(hipGetErrorString(err));
        }

        err = hipMemcpyAsync(dst, src, spec.length, spec.direction, stream);
        if (err != hipSuccess)
        {
            throw runtime_error(hipGetErrorString(err));
        }

        err = hipEventRecord(spec.timer->stopped, stream);
        if (err != hipSuccess)
        {
            throw runtime_error(hipGetErrorString(err));
        }
    }
}


// Wait for all streams to complete
static void syncStreams(const vector<TransferSpec>& transferSpecs)
{
    hipError_t err;

    for (const TransferSpec& spec : transferSpecs)
    {
        err = hipStreamSynchronize(*spec.stream);
        if (err != hipSuccess)
        {
            throw runtime_error(hipGetErrorString(err));
        }
    }
}


void runBandwidthTest(const vector<TransferSpec>& transferSpecs)
{
    hipError_t err;

    // Create timing events on the null stream
    TimerPtr globalTimer = createTimer();
    err = hipEventRecord(globalTimer->started);
    if (err != hipSuccess)
    {
        throw runtime_error(hipGetErrorString(err));
    }

    // Execute transfers
    try
    {
        fprintf(stdout, "Executing transfers..........");
        fflush(stdout);
        timeTransfers(transferSpecs);
        fprintf(stdout, "DONE\n");
        fflush(stdout);
    }
    catch (const runtime_error& e)
    {
        fprintf(stdout, "FAIL\n");
        fflush(stdout);
        throw e;
    }

    // Synchronize all streams
    try
    {
        fprintf(stdout, "Synchronizing streams........");
        fflush(stdout);

        syncStreams(transferSpecs);

        err = hipEventRecord(globalTimer->stopped);
        if (err != hipSuccess)
        {
            throw runtime_error(hipGetErrorString(err));
        }

        err = hipEventSynchronize(globalTimer->stopped);
        if (err != hipSuccess)
        {
            throw runtime_error(hipGetErrorString(err));
        }

        fprintf(stdout, "DONE\n");
        fflush(stdout);
    } 
    catch (const runtime_error& e)
    {
        fprintf(stdout, "FAIL\n");
        fflush(stdout);
        throw e;
    }


    // FIXME: Warn about low compute-capability here instead?

    // Print results
    fprintf(stdout, "\n");
    fprintf(stdout, "=====================================================================================\n");
    fprintf(stdout, " %2s   %-15s   %13s   %-8s   %-12s   %-10s\n",
            "ID", "Device name", "Transfer size", "Direction", "Time elapsed", "Bandwidth");
    fprintf(stdout, "-------------------------------------------------------------------------------------\n");
    fflush(stdout);

    size_t totalSize = 0;
    double aggrElapsed = .0;
    double timedElapsed = globalTimer->usecs();

    for (const TransferSpec& res : transferSpecs)
    {
        double elapsed = res.timer->usecs();
        double bandwidth = (double) res.length / elapsed;

        totalSize += res.length;
        aggrElapsed += elapsed;

        hipDeviceProp_t prop;
        err = hipGetDeviceProperties(&prop, res.device);
        if (err != hipSuccess)
        {
            prop.name[0] = 'E';
            prop.name[1] = 'R';
            prop.name[2] = 'R';
            prop.name[3] = '!';
            prop.name[4] = '\0';
        }

        fprintf(stdout, " %2d   %-15s   %13s    %8s   %9.0f µs    %10.2f MiB/s \n",
                res.device, 
                prop.name, 
                bytesToUnit(res.length).c_str(), 
                transferDirectionToString(res.direction).c_str(),
                elapsed,
                bandwidth
               );
        fflush(stdout);
    }
    fprintf(stdout, "=====================================================================================\n");

    fprintf(stdout, "\n");
    fprintf(stdout, "Aggregated total time      : %12.0f µs\n", aggrElapsed);
    fprintf(stdout, "Aggregated total bandwidth : %12.2f MiB/s\n", (double) totalSize / aggrElapsed);
    fprintf(stdout, "Estimated elapsed time     : %12.0f µs\n", timedElapsed);
    fprintf(stdout, "Timed total bandwidth      : %12.2f MiB/s\n", (double) totalSize / timedElapsed);
    fprintf(stdout, "\n");
    fflush(stdout);
}
